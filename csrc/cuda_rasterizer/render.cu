#include <hip/hip_runtime.h>
#include <stdint.h>

constexpr uint32_t WARP_SIZE = 32;
constexpr uint32_t BLOCK_X = 32;
constexpr uint32_t BLOCK_Y = 16;
constexpr uint32_t THREAD_X = 4;
constexpr uint32_t THREAD_Y = 4;

// Check keys to see if it is at the start/end of one tile's range in
// the full sorted list. If yes, write start/end of this tile.
// Run once per instanced (duplicated) Gaussian ID.
//识别排序后的键列表中每个tile范围的起始和结束位置
__global__ void identifyTileRanges(int L, uint64_t* point_list_keys, int2* ranges)  //L：排序后的键列表的长度  point_list_keys：key列表 输出ranges：存储每个tile范围开始和结束的位置 //64
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= L)
		return;

	// Read tile ID from key. Update start/end of tile range if at limit.
	uint64_t key = point_list_keys[idx];  //读取当前key，从总提取tile id 64
	uint32_t currtile = key >> 32; //32
	if (idx == 0)
		ranges[currtile].x = 0;  //如果当前索引是列表的第一个元素，将当前tile范围的起始位置设置为 0
	else   //否则，对于其他索引，检查当前tile是否与前一个tile相同
	{
		uint32_t prevtile = point_list_keys[idx - 1] >> 32; //32
		if (currtile != prevtile)    //如果当前tile与前一个tile不同，则更新前一个tile范围的结束位置和当前tile范围的起始位置
		{
			ranges[prevtile].y = idx;
			ranges[currtile].x = idx;
		}
	}
	if (idx == L - 1)
		ranges[currtile].y = L;
}

__forceinline__ __device__ void pixel_shader(float3& C, float& T, float2 pixf, float2 xy, float4 con_o, float3 rgb)
{
	float2 d = { xy.x - pixf.x, xy.y - pixf.y };
	//float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
	float power = con_o.w + con_o.x * d.x * d.x + con_o.z * d.y * d.y + con_o.y * d.x * d.y;
	float alpha;
	asm volatile("ex2.approx.ftz.f32 %0, %1;" : "=f"(alpha) : "f"(power));
	"\"";
	//alpha = min(0.99f, alpha);
	C.x += rgb.x * (alpha * T);
	C.y += rgb.y * (alpha * T);
	C.z += rgb.z * (alpha * T);
	T -= alpha * T;
}

__forceinline__ __device__ uint8_t encode(float x)
{
	return (uint8_t)min(max(0.0f, x * 255.0f), 255.0f);
}

__forceinline__ __device__ uint8_t write_color(uchar3* __restrict__ out_color,
	float3 bg_color, uint2 pix, int width, int height, float3 C, float T)
{
	if (pix.x < width && pix.y < height)
	{
		int pix_id = width * pix.y + pix.x;
		if (T < 0.0001f)
		{
			T = 0.0f;
		}
		out_color[pix_id].x = encode(C.x + T * bg_color.x);
		out_color[pix_id].y = encode(C.y + T * bg_color.y);
		out_color[pix_id].z = encode(C.z + T * bg_color.z);
	}
}

__global__ void renderCUDA(
	const int2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int width, int height, int horizontal_blocks,
	const float2* __restrict__ points_xy_image,
	const float3* __restrict__ features,
	const float* __restrict__ depths,
	const float4* __restrict__ conic_opacity,
	float3 bg_color,
	uchar3* __restrict__ out_color)
{
	int2 range = ranges[blockIdx.y * horizontal_blocks + blockIdx.x];
	uint32_t lane_id = threadIdx.y * blockDim.x + threadIdx.x;
	const void* data = nullptr;
	int scale = 0;
	switch (lane_id)
	{
	case 0:
		data = point_list;
		scale = 1;
		break;
	case 8:
		data = &points_xy_image->x;
		scale = 2;
		break;
	case 9:
		data = &points_xy_image->y;
		scale = 2;
		break;
	case 16:
		data = &features->x;
		scale = 3;
		break;
	case 17:
		data = &features->y;
		scale = 3;
		break;
	case 18:
		data = &features->z;
		scale = 3;
		break;
	case 24:
		data = &conic_opacity->x;
		scale = 4;
		break;
	case 25:
		data = &conic_opacity->y;
		scale = 4;
		break;
	case 26:
		data = &conic_opacity->z;
		scale = 4;
		break;
	case 27:
		data = &conic_opacity->w;
		scale = 4;
		break;
	}
	scale *= 4; // sizeof(int), sizeof(float)
	if (range.x >= range.y)
	{
		return;
	}
	int point_id = range.x;
	int coll_id = point_list[point_id];
	float2 xy = points_xy_image[coll_id];
	float3 rgb = features[coll_id];
	float4 con_o = conic_opacity[coll_id];
	coll_id = point_list[point_id + 1];
	if (lane_id == 0)
	{
		coll_id = point_id + 2;
	}

	// uint2 pix = { blockIdx.x * BLOCK_X + threadIdx.x, blockIdx.y * BLOCK_Y + threadIdx.y };
	uint2 pix[THREAD_Y][THREAD_X];
#pragma unroll
	for (uint32_t i = 0; i < THREAD_Y; i++)
	{
#pragma unroll
		for (uint32_t j = 0; j < THREAD_X; j++)
		{
			pix[i][j] = {
				blockIdx.x * BLOCK_X + threadIdx.x * THREAD_X + j,
				blockIdx.y * BLOCK_Y + threadIdx.y * THREAD_Y + i
			};
		}
	}

	// float2 pixf = { (float)pix.x, (float)pix.y };
	float2 pixf[THREAD_Y][THREAD_X];
#pragma unroll
	for (uint32_t i = 0; i < THREAD_Y; i++)
	{
#pragma unroll
		for (uint32_t j = 0; j < THREAD_X; j++)
		{
			pixf[i][j] = { (float)pix[i][j].x, (float)pix[i][j].y };
		}
	}

	float T[THREAD_Y][THREAD_X];
#pragma unroll
	for (uint32_t i = 0; i < THREAD_Y; i++)
	{
#pragma unroll
		for (uint32_t j = 0; j < THREAD_X; j++)
		{
			T[i][j] = 1.0f;
		}
	}
	float3 C[THREAD_Y][THREAD_X] = { 0.0f };
	bool done = true;
	float buf;
	coll_id *= scale;
	do {
		if (data != nullptr)
		{
			buf = *reinterpret_cast<const float*>(reinterpret_cast<const char*>(data) + coll_id);
		}

#pragma unroll
		for (uint32_t i = 0; i < THREAD_Y; i++)
		{
#pragma unroll
			for (uint32_t j = 0; j < THREAD_X; j++)
			{
				pixel_shader(C[i][j], T[i][j], pixf[i][j], xy, con_o, rgb);
			}
		}

		done = true;
#pragma unroll
		for (uint32_t i = 0; i < THREAD_Y; i++)
		{
#pragma unroll
			for (uint32_t j = 0; j < THREAD_X; j++)
			{
				done = done && T[i][j] < 0.0001f;
			}
		}
		coll_id = __shfl_sync(~0, __float_as_uint(buf), 0);
		if (lane_id == 0)
		{
			coll_id = point_id + 3;
		}
		xy = {
			__shfl_sync(~0, buf, 8),
			__shfl_sync(~0, buf, 9)
		};
		rgb = {
			__shfl_sync(~0, buf, 16),
			__shfl_sync(~0, buf, 17),
			__shfl_sync(~0, buf, 18)
		};
		con_o = {
			__shfl_sync(~0, buf, 24),
			__shfl_sync(~0, buf, 25),
			__shfl_sync(~0, buf, 26),
			__shfl_sync(~0, buf, 27)
		};
		coll_id *= scale;
	} while (__any_sync(~0, ++point_id < range.y && !done));
#pragma unroll
	for (uint32_t i = 0; i < THREAD_Y; i++)
	{
#pragma unroll
		for (uint32_t j = 0; j < THREAD_X; j++)
		{
			write_color(out_color, bg_color, pix[i][j], width, height, C[i][j], T[i][j]);
		}
	}
}


void render(int num_rendered,
	int width, int height,
	float* points_xy, float* depths, float* rgb, float* conic_opacity,
	uint64_t* gaussian_keys_sorted, uint32_t* gaussian_values_sorted,
	int* ranges,
	float3 bg_color, char* out_color)
{
	dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
	hipMemsetAsync(ranges, 0, sizeof(int2) * grid.x * grid.y);

	identifyTileRanges<<<(num_rendered + 255) / 256, 256>>>(
		num_rendered,
		gaussian_keys_sorted,
		(int2*)ranges);

	int horizontal_blocks = (width + BLOCK_X - 1) / BLOCK_X;
	renderCUDA<<<grid, dim3(8, 4, 1)>>>(
		(int2*)ranges,
		gaussian_values_sorted,
		width, height,
		horizontal_blocks,
		(float2*)points_xy,
		(float3*)rgb,
		depths,
		(float4*)conic_opacity,
		bg_color,
		(uchar3*)out_color);
}
